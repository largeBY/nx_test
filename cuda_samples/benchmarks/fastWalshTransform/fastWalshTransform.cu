/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/*
 * Walsh transforms belong to a class of generalized Fourier transformations.
 * They have applications in various fields of electrical engineering
 * and numeric theory. In this sample we demonstrate efficient implementation
 * of naturally-ordered Walsh transform
 * (also known as Walsh-Hadamard or Hadamard transform) in CUDA and its
 * particular application to dyadic convolution computation.
 * Refer to excellent Jorg Arndt's "Algorithms for Programmers" textbook
 * http://www.jjj.de/fxt/fxtbook.pdf (Chapter 22)
 *
 * Victor Podlozhnyuk (vpodlozhnyuk@nvidia.com)
 */



#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>


////////////////////////////////////////////////////////////////////////////////
// Reference CPU FWT
////////////////////////////////////////////////////////////////////////////////
extern"C" void fwtCPU(float *h_Output, float *h_Input, int log2N);
extern"C" void slowWTcpu(float *h_Output, float *h_Input, int log2N);
extern "C" void dyadicConvolutionCPU(
    float *h_Result,
    float *h_Data,
    float *h_Kernel,
    int log2dataN,
    int log2kernelN
);


////////////////////////////////////////////////////////////////////////////////
// GPU FWT
////////////////////////////////////////////////////////////////////////////////
#include "fastWalshTransform_kernel.cuh"



////////////////////////////////////////////////////////////////////////////////
// Data configuration
////////////////////////////////////////////////////////////////////////////////
const int log2Kernel = 7;
const   int log2Data = 23;

const int   dataN = 1 << log2Data;
const int kernelN = 1 << log2Kernel;

const int   DATA_SIZE = dataN   * sizeof(float);
const int KERNEL_SIZE = kernelN * sizeof(float);

const double NOPS = 3.0 * (double)dataN * (double)log2Data / 2.0;



////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char *argv[])
{
    float *h_Data,
          *h_Kernel,
          *h_ResultCPU,
          *h_ResultGPU;

    float *d_Data,
          *d_Kernel;

    double gpuTime;

    StopWatchInterface *hTimer = NULL;
    int i;
    int rep;

    printf("%s Starting...\n\n", argv[0]);

    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    findCudaDevice(argc, (const char **)argv);

    sdkCreateTimer(&hTimer);

    printf("Initializing data...\n");
    printf("...allocating CPU memory\n");
    h_Kernel    = (float *)malloc(KERNEL_SIZE);
    h_Data      = (float *)malloc(DATA_SIZE);
    h_ResultCPU = (float *)malloc(DATA_SIZE);
    h_ResultGPU = (float *)malloc(DATA_SIZE);
    printf("...allocating GPU memory\n");
    checkCudaErrors(hipMalloc((void **)&d_Kernel, DATA_SIZE));
    checkCudaErrors(hipMalloc((void **)&d_Data,   DATA_SIZE));

    printf("...generating data\n");
    printf("Data length: %i; kernel length: %i\n", dataN, kernelN);
    srand(2007);

    for (i = 0; i < kernelN; i++)
    {
        h_Kernel[i] = (float)rand() / (float)RAND_MAX;
    }

    for (i = 0; i < dataN; i++)
    {
        h_Data[i] = (float)rand() / (float)RAND_MAX;
    }

    checkCudaErrors(hipMemset(d_Kernel, 0, DATA_SIZE));
    checkCudaErrors(hipMemcpy(d_Kernel, h_Kernel, KERNEL_SIZE, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_Data,   h_Data,     DATA_SIZE, hipMemcpyHostToDevice));

    printf("Running GPU dyadic convolution using Fast Walsh Transform...\n");
    checkCudaErrors(hipDeviceSynchronize());

    time_t t;
    struct tm * lt;
    time(&t);
    lt = localtime(&t);
    printf("%d-%d-%d %d:%d:%d\n" ,lt->tm_year+1900, lt->tm_mon+1, lt->tm_mday, lt->tm_hour, lt->tm_min, lt->tm_sec);

    sdkResetTimer(&hTimer);
    sdkStartTimer(&hTimer);
    for (rep = 0; rep < 54000; rep++)
    {
        fwtBatchGPU(d_Data, 1, log2Data);
        fwtBatchGPU(d_Kernel, 1, log2Data);
        modulateGPU(d_Data, d_Kernel, dataN);
        fwtBatchGPU(d_Data, 1, log2Data);
        checkCudaErrors(hipDeviceSynchronize());
    }
    sdkStopTimer(&hTimer);
    gpuTime = sdkGetTimerValue(&hTimer);
    time(&t);
    lt = localtime(&t);
    printf("%d-%d-%d %d:%d:%d\n" ,lt->tm_year+1900, lt->tm_mon+1, lt->tm_mday, lt->tm_hour, lt->tm_min, lt->tm_sec);
    printf("GPU time: %f ms; GOP/s: %f\n", gpuTime, 30000 * NOPS / (gpuTime * 0.001 * 1E+9));


    printf("Shutting down...\n");
    sdkDeleteTimer(&hTimer);
    checkCudaErrors(hipFree(d_Data));
    checkCudaErrors(hipFree(d_Kernel));
    free(h_ResultGPU);
    free(h_ResultCPU);
    free(h_Data);
    free(h_Kernel);

}
